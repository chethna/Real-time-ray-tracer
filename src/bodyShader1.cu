#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
//#include <optix_world.h>
#include "commonStructs.h"
#include "helpers.h"

using namespace optix;

rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(int,          max_depth, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(float,        importance_cutoff, , );
rtDeclareVariable(float3,       cutoff_color, , );
rtDeclareVariable(float,        fresnel_exponent, , );
rtDeclareVariable(float,        fresnel_minimum, , );
rtDeclareVariable(float,        fresnel_maximum, , );
rtDeclareVariable(float,        refraction_index, , );
rtDeclareVariable(int,          refraction_maxdepth, , );
rtDeclareVariable(int,          reflection_maxdepth, , );
rtDeclareVariable(float3,       refraction_color, , );
rtDeclareVariable(float3,       reflection_color, , );
rtDeclareVariable(float3,       extinction_constant, , );

rtBuffer<BasicLight>                 lights;

rtTextureSampler<float4, 2> diffusemap;
rtTextureSampler<float4, 2> transpmap;
rtTextureSampler<float4, 2> c0map;
rtTextureSampler<float4, 2> c1map;
//rtTextureSampler<float4, 2> bumpmap;

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

struct PerRayData_shadow
{
  float3 attenuation;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
// -----------------------------------------------------------------------------

static __device__ __inline__ float3 TraceRay(float3 origin, float3 direction, int depth, float importance )
{
  optix::Ray ray = optix::make_Ray( origin, direction, radiance_ray_type, 0.0f, RT_DEFAULT_MAX );
  PerRayData_radiance prd;
  prd.depth = depth;
  prd.importance = importance;

  rtTrace( top_object, ray, prd );
  return prd.result;
}

static __device__ __inline__ float3 exp( const float3& x )
{
  return make_float3(exp(x.x), exp(x.y), exp(x.z));
}

// -----------------------------------------------------------------------------

RT_PROGRAM void closest_hit_radiance()
{
    // intersection vectors
    const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
    float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    const float3 fhp = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);
    const float3 bhp = rtTransformPoint(RT_OBJECT_TO_WORLD, back_hit_point);

    const float3 i = ray.direction;                                            // incident direction
          float3 t;                                                            // transmission direction
          float3 r;                                                            // reflection direction

    float reflection = 1.0f;
    float3 result = make_float3(0.0f);

    const int depth = prd_radiance.depth;

    float3 beer_attenuation;

    if(dot(n, ray.direction) > 0) {
      // Beer's law attenuation
      beer_attenuation = exp(extinction_constant * t_hit);
    } else {
      beer_attenuation = make_float3(1);
    }

    // refraction
    if (depth < min(refraction_maxdepth, max_depth))
    {
      if ( refract(t, i, n, refraction_index) )
      {
        // check for external or internal reflection
        float cos_theta = dot(i, n);
        if (cos_theta < 0.0f)
          cos_theta = -cos_theta;
        else
          cos_theta = dot(t, n);

        reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

        float importance = prd_radiance.importance * (1.0f-reflection) * optix::luminance( refraction_color * beer_attenuation );
        float3 color = cutoff_color;
        if ( importance > importance_cutoff ) {
          color = TraceRay(bhp, t, depth+1, importance);
        }
        result += (1.0f - reflection) * refraction_color * color;
      }
      // else TIR
    } // else reflection==1 so refraction has 0 weight

    // reflection
    float3 color = cutoff_color;
    if (depth < min(reflection_maxdepth, max_depth))
    {
      r = reflect(i, n);

      float importance = prd_radiance.importance * reflection * optix::luminance( reflection_color * beer_attenuation );
      if ( importance > importance_cutoff ) {
        color = TraceRay( fhp, r, depth+1, importance );
      }
    }
    result += reflection * reflection_color * color;

    result = result * beer_attenuation;

    float3 resultReflect = make_float3(1.0f,1.0f,1.0f);
    resultReflect = resultReflect*result;
    //prd_radiance.result = result;


  /*
   *
   * result += reflection * reflection_color * color;

  result = result * beer_attenuation;

  prd_radiance.result = result;
   *
   */

  float tnew  = dot(-normalize(ray.direction),normalize(n));
  tnew= pow(tnew,4.0f);
  //float3 c0Color = make_float3( tex2D(c0map, texcoord.x, texcoord.y ));
  //float3 c1Color = make_float3( tex2D(c1map, texcoord.x, texcoord.y ));
  float3 c0Color = make_float3(222.0f/255.0f,128.0f/255.0f,146.0f/255.0f);
  float3 c1Color = make_float3(216.0f/255.0f,186.0f/255.0f,186.0f/255.0f);
  result = (1-tnew)*c0Color + c1Color*tnew;
  prd_radiance.result = result;

  float3 p_normal = faceforward( n, -ray.direction, world_geometric_normal );
  float3 R = reflect( ray.direction, p_normal );
  float3 hit_point = ray.origin + t_hit * ray.direction;

  float3 csColor = make_float3(1.0f,1.0f,1.0f);
  unsigned int num_lights = lights.size();
  for(int i = 0; i < num_lights; ++i) {
    BasicLight light = lights[i];
    //float Ldist = optix::length(light.pos - hit_point);
    float3 L = optix::normalize(light.pos - hit_point);
    float snew =  dot(R,L);
    if(snew > 0.99f){
        snew = 1.0f;
    }
    else{
        snew = 0.0f;
    }
    result = (1-snew)*result + csColor*snew;
    //float nDl = optix::dot( p_normal, L);

  }

  float bnew = 1 - tnew;
  float3 cbColor = make_float3(124.0f/255.0f,100.0f/255.0f,113.0f/255.0f);
  if(bnew>0.995f){
      bnew = 1.0f;
  }
  else{
      bnew = 0.0f;
  }

  result = cbColor*bnew + result*(1-bnew);

  float T = make_float3(1.0f,1.0f,1.0f).x;
          //make_float3( tex2D(transpmap, texcoord.x, texcoord.y )).x;

  if(bnew == 0.0f){
      //result = T*result + (1 - T)*resultReflect;
  }

  prd_radiance.result = result;

}


// -----------------------------------------------------------------------------

//
// Attenuates shadow rays for shadowing transparent objects
//
rtDeclareVariable(float3, shadow_attenuation, , );

RT_PROGRAM void any_hit_shadow()
{
  float3 world_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float nDi = fabs(dot(world_normal, ray.direction));

  prd_shadow.attenuation *= 1-fresnel_schlick(nDi, 5, 1-shadow_attenuation, make_float3(1));
  if(optix::luminance(prd_shadow.attenuation) < importance_cutoff)
    rtTerminateRay();
  else
    rtIgnoreIntersection();
}
