#include "hip/hip_runtime.h"
// Environment map background

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtTextureSampler<float4, 2>      diffuse_map; // Corresponds to OBJ mtl params

RT_PROGRAM void closest_hit_radiance()
{
  //const float3 uv = texcoord;
  float3 Kd = make_float3( tex2D( diffuse_map, texcoord.x, texcoord.y ) );
  prd_radiance.result = Kd;//make_float3( tex2D(tex_map, uv.x, uv.y) );
  //rtPrintf( "Environment texture color: %d, %d, %d!\n", prd_radiance.result.x, prd_radiance.result.y, prd_radiance.result.z );
}



/*#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

// surface textures
rtTextureSampler<float, 2>     kd_map;

struct PerRayData_tex
{
  float3 result;
};

rtDeclareVariable(PerRayData_tex, prd, rtPayload, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );




RT_PROGRAM void closest_hit_radiance()
{
  const float3 uv = texcoord;

  prd.result = make_float3(1.0f, 1.0f, 1.0f);//make_float3( tex2D( kd_map, uv.x, uv.y ) );
}
*/

